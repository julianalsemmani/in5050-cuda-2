#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>

#include "me.h"
#include "tables.h"

struct mv_data
{
  int sad;
  int mv_x;
  int mv_y;
};

__device__ static void sad_block_8x8(uint8_t *block1, uint8_t *block2, int stride, int *result, int u, int v)
{
  int difference = abs(block2[v * stride + u] - block1[v * stride + u]);
  atomicAdd(result, difference);
}

/* Motion estimation for 8x8 block */
__global__ static void me_block_8x8(struct c63_common *cm, struct macroblock *mb_gpu, uint8_t *orig, uint8_t *ref, int color_component)
{
  int mb_x = blockIdx.x;
  int mb_y = blockIdx.y;
  struct macroblock *mb = &mb_gpu[mb_y*cm->padw[color_component]/8+mb_x];

  int range = cm->me_search_range;

  /* Quarter resolution for chroma channels. */
  if (color_component > 0) { range /= 2; }


  int left = mb_x * 8 - range;
  int top = mb_y * 8 - range;
  int right = mb_x * 8 + range;
  int bottom = mb_y * 8 + range;

  int w = cm->padw[color_component];
  int h = cm->padh[color_component];

  /* Make sure we are within bounds of reference frame. TODO: Support partial
     frame bounds. */
  if (left < 0) { left = 0; }
  if (top < 0) { top = 0; }
  if (right > (w - 8)) { right = w - 8; }
  if (bottom > (h - 8)) { bottom = h - 8; }

  int x, y;

  int mx = mb_x * 8;
  int my = mb_y * 8;

  int best_sad = INT_MAX;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      __shared__ int sad;
      sad = 0;

      __syncthreads();

      sad_block_8x8(orig + my*w+mx, ref + y*w+x, w, &sad, threadIdx.x, threadIdx.y);

      __syncthreads();

      if (sad < best_sad)
      {
        mb->mv_x = x - mx;
        mb->mv_y = y - my;
        best_sad = sad;
      }
    }
  }

  /* Here, there should be a threshold on SAD that checks if the motion vector
     is cheaper than intraprediction. We always assume MV to be beneficial */

  /* printf("Using motion vector (%d, %d) with SAD %d\n", mb->mv_x, mb->mv_y,
     best_sad); */

  mb->use_mv = 1;
}

void c63_motion_estimate(struct c63_common *cm)
{
  /* Compare this frame with previous reconstructed frame */
  int mb_x, mb_y;

  struct c63_common *cm_gpu;
  struct macroblock *mb_Y, *mb_U, *mb_V;

  hipMalloc((void **)&cm_gpu, sizeof(struct c63_common));

  hipMalloc((void **)&mb_Y, sizeof(struct macroblock)*(cm->mb_rows)*(cm->mb_cols));
  hipMalloc((void **)&mb_U, sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2));
  hipMalloc((void **)&mb_V, sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2));

  hipMemcpy(cm_gpu, cm, sizeof(struct c63_common), hipMemcpyHostToDevice);

  hipMemcpy(mb_Y, cm->curframe->mbs[Y_COMPONENT], sizeof(struct macroblock)*(cm->mb_rows)*(cm->mb_cols), hipMemcpyHostToDevice);
  hipMemcpy(mb_U, cm->curframe->mbs[U_COMPONENT], sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2), hipMemcpyHostToDevice);
  hipMemcpy(mb_V, cm->curframe->mbs[V_COMPONENT], sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2), hipMemcpyHostToDevice);

  uint8_t *orig_Y, *recons_Y;
  hipMalloc((void **)&orig_Y, sizeof(uint8_t)*cm->padw[Y_COMPONENT]*cm->padh[Y_COMPONENT]);
  hipMalloc((void **)&recons_Y, sizeof(uint8_t)*cm->padw[Y_COMPONENT]*cm->padh[Y_COMPONENT]);

  uint8_t *orig_U, *recons_U;
  hipMalloc((void **)&orig_U, sizeof(uint8_t)*cm->padw[U_COMPONENT]*cm->padh[U_COMPONENT]);
  hipMalloc((void **)&recons_U, sizeof(uint8_t)*cm->padw[U_COMPONENT]*cm->padh[U_COMPONENT]);

  uint8_t *orig_V, *recons_V;
  hipMalloc((void **)&orig_V, sizeof(uint8_t)*cm->padw[V_COMPONENT]*cm->padh[V_COMPONENT]);
  hipMalloc((void **)&recons_V, sizeof(uint8_t)*cm->padw[V_COMPONENT]*cm->padh[V_COMPONENT]);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipMemcpy(orig_Y, cm->curframe->orig->Y, sizeof(uint8_t)*cm->padw[Y_COMPONENT]*cm->padh[Y_COMPONENT], hipMemcpyHostToDevice);
  hipMemcpy(orig_U, cm->curframe->orig->U, sizeof(uint8_t)*cm->padw[U_COMPONENT]*cm->padh[U_COMPONENT], hipMemcpyHostToDevice);
  hipMemcpy(orig_V, cm->curframe->orig->V, sizeof(uint8_t)*cm->padw[V_COMPONENT]*cm->padh[V_COMPONENT], hipMemcpyHostToDevice);
  hipMemcpy(recons_Y, cm->curframe->recons->Y, sizeof(uint8_t)*cm->padw[Y_COMPONENT]*cm->padh[Y_COMPONENT], hipMemcpyHostToDevice);
  hipMemcpy(recons_U, cm->curframe->recons->U, sizeof(uint8_t)*cm->padw[U_COMPONENT]*cm->padh[U_COMPONENT], hipMemcpyHostToDevice);
  hipMemcpy(recons_V, cm->curframe->recons->V, sizeof(uint8_t)*cm->padw[V_COMPONENT]*cm->padh[V_COMPONENT], hipMemcpyHostToDevice);

  dim3 gridDim(cm->mb_cols, cm->mb_rows);
  dim3 blockDim(8, 8);

  /* Luma */
  me_block_8x8<<<gridDim, blockDim>>>(cm_gpu, mb_Y, orig_Y, recons_Y, Y_COMPONENT);

  /* Chroma */
  me_block_8x8<<<gridDim, blockDim>>>(cm_gpu, mb_U, orig_U, recons_U, U_COMPONENT);

  me_block_8x8<<<gridDim, blockDim>>>(cm_gpu, mb_V, orig_V, recons_V, V_COMPONENT);

  hipDeviceSynchronize();

  hipMemcpy(cm->curframe->mbs[Y_COMPONENT], mb_Y, sizeof(struct macroblock)*(cm->mb_rows)*(cm->mb_cols), hipMemcpyDeviceToHost);
  hipMemcpy(cm->curframe->mbs[U_COMPONENT], mb_U, sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2), hipMemcpyDeviceToHost);
  hipMemcpy(cm->curframe->mbs[V_COMPONENT], mb_V, sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2), hipMemcpyDeviceToHost);

  hipFree(orig_Y);
  hipFree(recons_Y);
  hipFree(orig_U);
  hipFree(recons_U);
  hipFree(orig_V);
  hipFree(recons_V);
  hipFree(cm_gpu);
  hipFree(mb_Y);
  hipFree(mb_U);
  hipFree(mb_V);
}

/* Motion compensation for 8x8 block */
static void mc_block_8x8(struct c63_common *cm, int mb_x, int mb_y,
    uint8_t *predicted, uint8_t *ref, int color_component)
{
  struct macroblock *mb =
    &cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];

  if (!mb->use_mv) { return; }

  int left = mb_x * 8;
  int top = mb_y * 8;
  int right = left + 8;
  int bottom = top + 8;

  int w = cm->padw[color_component];

  /* Copy block from ref mandated by MV */
  int x, y;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      predicted[y*w+x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
    }
  }
}

void c63_motion_compensate(struct c63_common *cm)
{
  int mb_x, mb_y;

  /* Luma */
  for (mb_y = 0; mb_y < cm->mb_rows; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->Y,
          cm->refframe->recons->Y, Y_COMPONENT);
    }
  }

  /* Chroma */
  for (mb_y = 0; mb_y < cm->mb_rows / 2; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols / 2; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->U,
          cm->refframe->recons->U, U_COMPONENT);
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->V,
          cm->refframe->recons->V, V_COMPONENT);
    }
  }
}